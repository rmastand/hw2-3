#include "hip/hip_runtime.h"
#include "common.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <thrust/scan.h>
#include <cuda/atomic>
#include <cuda/atomic>


#define NUM_THREADS 256

// Put any static global variables here that you will use throughout the simulation.
int blks;

static int NUM_BLOCKS;
int tot_num_bins;

// Initialize arrays for particle ids and bin ids
int* bin_ids;
int* sorted_particles;
// Array to store how many particles in a bin have been added to sorted_particles
int* how_many_filled;

__device__ void apply_force_gpu(particle_t& particle, particle_t& neighbor) {
    double dx = neighbor.x - particle.x;
    double dy = neighbor.y - particle.y;
    double r2 = dx * dx + dy * dy;
    if (r2 > cutoff * cutoff)
        return;
    // r2 = fmax( r2, min_r*min_r );
    r2 = (r2 > min_r * min_r) ? r2 : min_r * min_r;
    double r = sqrt(r2);

    //
    //  very simple short-range repulsive force
    //
    double coef = (1 - cutoff / r) / r2 / mass;
    particle.ax += coef * dx;
    particle.ay += coef * dy;
}

__global__ void compute_forces_gpu(particle_t* parts, int* sorted_particles, int* bin_ids, int num_parts, double size, int NUM_BLOCKS) {
    // Get thread (particle) ID
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid >= num_parts)
        return;

    parts[tid].ax = parts[tid].ay = 0;

     // Get what row and column the particle would be in, with padding
    int dx = (parts[tid].x * NUM_BLOCKS / size) + 1;
    int dy = (parts[tid].y * NUM_BLOCKS / size) + 1;
    // Get the bin id of the particle
    //int my_bin_id = dx + (NUM_BLOCKS+2)*dy;

    // Iterate through the 3x3 neighboring bins
    for (int m = -1; m <= 1; m++) {
        for (int n = -1; n <=1; n++) {

            // Get the bin_id of the neighboring bin
            int their_bin_id = dx + m + (NUM_BLOCKS+2)*(dy+n);

            // Iterate through all the particles in their_bin_id
            int their_bin_id_start = bin_ids[their_bin_id - 1];
            int next_bin_id_start = bin_ids[their_bin_id];

            for (int j = their_bin_id_start; j < next_bin_id_start; j++){

                int particle_j_id = sorted_particles[j];
                apply_force_gpu(parts[tid], parts[particle_j_id]);
            }
        }
    }
}

__global__ void move_gpu(particle_t* particles, int num_parts, double size) {

    // Get thread (particle) ID
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid >= num_parts)
        return;

    particle_t* p = &particles[tid];
    //
    //  slightly simplified Velocity Verlet integration
    //  conserves energy better than explicit Euler method
    //
    p->vx += p->ax * dt;
    p->vy += p->ay * dt;
    p->x += p->vx * dt;
    p->y += p->vy * dt;

    //
    //  bounce from walls
    //
    while (p->x < 0 || p->x > size) {
        p->x = p->x < 0 ? -(p->x) : 2 * size - p->x;
        p->vx = -(p->vx);
    }
    while (p->y < 0 || p->y > size) {
        p->y = p->y < 0 ? -(p->y) : 2 * size - p->y;
        p->vy = -(p->vy);
    }
}



__global__ void initialize_array_zeros_gpu(int* array, int array_size) {

    // Get thread (particle) ID
    int tid = threadIdx.x + blockIdx.x * blockDim.x;

    // Initialize the array with -1
    if (tid >= array_size) {
        return; 
    }
        array[tid] = 0;
    }


__global__ void count_particles_per_bin(particle_t* parts, int* bin_ids, int num_parts, double size, int NUM_BLOCKS) {

    // Get thread (particle) ID
    int tid = threadIdx.x + blockIdx.x * blockDim.x;

    if (tid >= num_parts) {
        return; 
    }
    else {
        // Get what row and column the particle would be in, with padding
        int dx = (parts[tid].x * NUM_BLOCKS / size) + 1;
        int dy = (parts[tid].y * NUM_BLOCKS / size) + 1;
        // Get the bin id of the particle
        int bin_id = dx + (NUM_BLOCKS+2)*dy;

        // Increment the relevant bin_id
        atomicAdd(&bin_ids[bin_id], 1);
    }

}

__global__ void bin_particles(particle_t* parts, int* sorted_particles, int* bin_ids, int* how_many_filled, int num_parts, double size, int NUM_BLOCKS) {

    // Get thread (particle) ID
    int tid = threadIdx.x + blockIdx.x * blockDim.x;

    if (tid >= num_parts) {
        return; 
    }
    else {
        // Get what row and column the particle would be in, with padding
        int dx = (parts[tid].x * NUM_BLOCKS / size) + 1;
        int dy = (parts[tid].y * NUM_BLOCKS / size) + 1;
        // Get the bin id of the particle
        int bin_id = dx + (NUM_BLOCKS+2)*dy;

        // Get the id of where the particle will be stored in 
            // The particles for that bin start at position in array bin_ids[bin_id - 1] in sorted_particles
            // This particle goes to bin_ids[bin_id - 1] + loc_index
            // get loc_index from an atomic fetch_add in how_many_filled[bin_id]

        int bin_index_start = bin_ids[bin_id - 1]; // Don't need to worry about bin_id = 0 due to zero-padding
        int loc_index = how_many_filled[bin_id].fetch_add(1, cuda::memory_order_relaxed);

        sorted_particles[bin_index_start + loc_index] = tid; //should this just be tid?  
    }
}


__global__ void initialize_5(int* array, int array_size) {

    // Get thread (particle) ID
    int tid = threadIdx.x + blockIdx.x * blockDim.x;

    // Initialize the array with -1
    if (tid >= array_size) {
        return; 
    }
        array[tid] = 5;
    }


void init_simulation(particle_t* parts, int num_parts, double size) {
    // You can use this space to initialize data objects that you may need
    // This function will be called once before the algorithm begins
    // parts live in GPU memory
    // Do not do any particle simulation here

    blks = (num_parts + NUM_THREADS - 1) / NUM_THREADS;

    // num blocks in either x or y direction (+2 in each dimension for padding)
    NUM_BLOCKS = size/cutoff;
    tot_num_bins = (NUM_BLOCKS+2)*(NUM_BLOCKS+2);

    hipMalloc((void**)&bin_ids, tot_num_bins * sizeof(int));
    hipMalloc((void**)&how_many_filled, num_parts * sizeof(int));
    hipMalloc((void**)&sorted_particles, num_parts * sizeof(int));
    
}

void simulate_one_step(particle_t* parts, int num_parts, double size) {
    // parts live in GPU memory

    // Initialize the array of bins_ids to have all 0's
    initialize_array_zeros_gpu<<<blks, NUM_THREADS>>>(bin_ids, tot_num_bins);
    initialize_array_zeros_gpu<<<blks, NUM_THREADS>>>(how_many_filled, tot_num_bins);

    // count the number of particles per bin
    count_particles_per_bin<<<blks, NUM_THREADS>>>(parts, bin_ids, num_parts, size, NUM_BLOCKS);

    int* bin_ids_cpu = (int*) malloc(tot_num_bins * sizeof(int));
    hipMemcpy(bin_ids_cpu, bin_ids, tot_num_bins * sizeof(int), hipMemcpyDeviceToHost);

    // prefix sum bin_ids into bin_counts
    thrust::inclusive_scan(thrust::device, bin_ids, bin_ids + tot_num_bins, bin_ids);
   
    // HORRIBLE NAMING but from this point, bin_ids is bin_counts
    // The number of particles in bin_i is bin_counts[i] - bin_counts[i-1]

    // test
    int* bin_counts_cpu = (int*) malloc(tot_num_bins * sizeof(int));
    int* part_links_cpu = (int*) malloc(num_parts * sizeof(int));

    hipMemcpy(bin_counts_cpu, bin_ids, tot_num_bins * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(part_links_cpu, sorted_particles, num_parts * sizeof(int), hipMemcpyDeviceToHost);
   
    for (int p = 0; p < tot_num_bins; p++) {
            std::cout << "testing bins " << p << " " <<bin_ids_cpu[p] <<  " " << " " << bin_counts_cpu[p] << std::endl;
    }

    // Add particles to separate array starting from bin idx
    bin_particles<<<blks, NUM_THREADS>>>(parts, sorted_particles, bin_ids, how_many_filled, num_parts, size, NUM_BLOCKS);

    // Compute forces
    //compute_forces_gpu<<<blks, NUM_THREADS>>>(parts, num_parts);

    // Move particles
    //move_gpu<<<blks, NUM_THREADS>>>(parts, num_parts, size);
}
